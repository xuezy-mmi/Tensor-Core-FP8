#include <cassert>
#include <chrono>
#include <cstdint>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <time.h>
#include <type_traits>
#include <vector>
#include <stdio.h>
#include <mma.h>
#include <float.h>

using namespace nvcuda;
typedef __hip_fp8_e5m2_fnuz e5m2;
typedef __hip_fp8_e4m3_fnuz e4m3;
// static constexpr int BLOCKM = 128;
// static constexpr int BLOCKN = 128;
// static constexpr int BLOCKK = 32;

// #define HOST_DEVICE __forceinline__ __host__ __device__
// #define DEVICE __forceinline__ __device__

namespace gemm{

//kernel1////////////////e4m3 * e4m3 = f32//////////////////
__global__ void GEMM_e4m3_e4m3_o32_stage2_row_col(
    int M, int K, int N,
    const e4m3 * __restrict__ A_Value,
    const e4m3 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    // thread num = 128
    constexpr int Block_M = 128;
    constexpr int Block_K = 64;
    constexpr int Block_N = 128;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int lane_id = threadIdx.x % 32;
    int wid = tid / 32;// 128 / 32 = 4

    if(bx >= M / Block_M || by >= N / Block_N){
        return;
    }
    const int APAD = 0;
    const int BPAD = 0;

    // extern __shared__ float4 smem[];
    const int smem_a_offset = Block_M * (Block_K + APAD);
    const int smem_b_offset = Block_N * (Block_K + BPAD);
    // const int smem_size_float4 = 2 * (smem_a_offset + smem_b_offset) / 16;
    __shared__ e4m3 smem_a[smem_a_offset * 2];
    __shared__ e4m3 smem_b[smem_b_offset * 2];
    // extern __shared__ float4 smem[];
    // e4m3 * smem_a = reinterpret_cast<e4m3 *>(smem);
    // e4m3 * smem_b = reinterpret_cast<e4m3 *>(smem_a + 2 * smem_a_offset);

    float4 matrix_a_fragment[8];// 8 float4 = 32 reg
    float4 matrix_b_fragment[8];// 8 float4 = 32 reg
    float output_fragment[128];// 4(m) * 8(n) * 4(one tile) = 128 float

    int smem_a_m = (tid / 4) * 4;
    int smem_a_k = (tid % 4) * 16;
    int smem_b_k = (tid % 4) * 16;
    int smem_b_n = (tid / 4) * 4;
    int smem_a_base_addr = __cvta_generic_to_shared(smem_a);
    int smem_b_base_addr = __cvta_generic_to_shared(smem_b);
    // block_size = 128 * 64 
    // thred num = 128 
    // each thread load = 128 * 64 / 128 = 64 fp8 = 4 float4
    int smem_a_addr0 = smem_a_base_addr + (smem_a_m * (Block_K + APAD) + smem_a_k) * sizeof(char);
    int smem_a_addr1 = smem_a_addr0 + 1 * (Block_K + APAD) * sizeof(char);
    int smem_a_addr2 = smem_a_addr0 + 2 * (Block_K + APAD) * sizeof(char);
    int smem_a_addr3 = smem_a_addr0 + 3 * (Block_K + APAD) * sizeof(char);
    
    int smem_b_addr0 = smem_b_base_addr + (smem_b_n * (Block_K + BPAD) + smem_b_k) * sizeof(char);
    int smem_b_addr1 = smem_b_addr0 + 1 * (Block_K + BPAD) * sizeof(char);
    int smem_b_addr2 = smem_b_addr0 + 2 * (Block_K + BPAD) * sizeof(char);
    int smem_b_addr3 = smem_b_addr0 + 3 * (Block_K + BPAD) * sizeof(char);

    int gmem_a_m = bx * Block_M + smem_a_m;
    int gmem_a_k = smem_a_k;
    int gmem_b_k = smem_b_k;
    int gmem_b_n = by * Block_N + smem_b_n;

    int gmem_a_addr = gmem_a_m * K + gmem_a_k;
    int gmem_b_addr = gmem_b_n * K + gmem_b_k;

    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr0), "l"(&A_Value[gmem_a_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr1), "l"(&A_Value[gmem_a_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr2), "l"(&A_Value[gmem_a_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr3), "l"(&A_Value[gmem_a_addr + 3 * K]));

    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr0), "l"(&B_Value[gmem_b_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr1), "l"(&B_Value[gmem_b_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr2), "l"(&B_Value[gmem_b_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr3), "l"(&B_Value[gmem_b_addr + 3 * K]));

    asm ("cp.async.commit_group;\n" ::);
    asm ("cp.async.wait_group 0;\n" ::);
    __syncthreads();

    int warp_x = wid % 2;//0 1 0 1
    int warp_y = wid / 2;//0 0 1 1
    #pragma unroll 32
    for(int bk = 1; bk < K/Block_K; bk++){
        int sel_com = (bk % 2) ^ 1; // 0 1 0 1
        int sel_mem = bk % 2; // 1 0 1 0
        gmem_a_addr += Block_K;
        gmem_b_addr += Block_K;
        
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr0 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr1 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 1 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr2 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 2 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr3 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 3 * K]));

        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr0 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr1 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 1 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr2 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 2 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr3 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 3 * K]));
        
        float4 * smem_a_sel = reinterpret_cast<float4 *>(smem_a + sel_com * smem_a_offset + warp_x * 64 * (Block_K+APAD));
        float4 * smem_b_sel = reinterpret_cast<float4 *>(smem_b + sel_com * smem_b_offset + warp_y * 64 * (Block_K+BPAD));
        matrix_a_fragment[0] = *(smem_a_sel + (lane_id % 4) + (int)(lane_id / 4 + 0 ) * (4+0));
        matrix_a_fragment[1] = *(smem_a_sel + (lane_id % 4) + (int)(lane_id / 4 + 8 ) * (4+0));
        matrix_a_fragment[2] = *(smem_a_sel + (lane_id % 4) + (int)(lane_id / 4 + 16) * (4+0));
        matrix_a_fragment[3] = *(smem_a_sel + (lane_id % 4) + (int)(lane_id / 4 + 24) * (4+0));
        matrix_a_fragment[4] = *(smem_a_sel + (lane_id % 4) + (int)(lane_id / 4 + 32) * (4+0));
        matrix_a_fragment[5] = *(smem_a_sel + (lane_id % 4) + (int)(lane_id / 4 + 40) * (4+0));
        matrix_a_fragment[6] = *(smem_a_sel + (lane_id % 4) + (int)(lane_id / 4 + 48) * (4+0));
        matrix_a_fragment[7] = *(smem_a_sel + (lane_id % 4) + (int)(lane_id / 4 + 56) * (4+0));
        matrix_b_fragment[0] = *(smem_b_sel + (lane_id % 4) + (int)(lane_id / 4 + 0 ) * (4+0));
        matrix_b_fragment[1] = *(smem_b_sel + (lane_id % 4) + (int)(lane_id / 4 + 8 ) * (4+0));
        matrix_b_fragment[2] = *(smem_b_sel + (lane_id % 4) + (int)(lane_id / 4 + 16) * (4+0));
        matrix_b_fragment[3] = *(smem_b_sel + (lane_id % 4) + (int)(lane_id / 4 + 24) * (4+0));
        matrix_b_fragment[4] = *(smem_b_sel + (lane_id % 4) + (int)(lane_id / 4 + 32) * (4+0));
        matrix_b_fragment[5] = *(smem_b_sel + (lane_id % 4) + (int)(lane_id / 4 + 40) * (4+0));
        matrix_b_fragment[6] = *(smem_b_sel + (lane_id % 4) + (int)(lane_id / 4 + 48) * (4+0));
        matrix_b_fragment[7] = *(smem_b_sel + (lane_id % 4) + (int)(lane_id / 4 + 56) * (4+0));

        int * a_fragment_int = reinterpret_cast<int *>(matrix_a_fragment);// 8 float4 --> 32 reg
        int * b_fragment_int = reinterpret_cast<int *>(matrix_b_fragment);// 8 float4 --> 32 reg

        #pragma unroll
        for(int i = 0; i < 4; i++){
            #pragma unroll
            for(int j = 0; j < 8; j++){
                asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 \t"
                    "{%0, %1, %2, %3}, \t"
                    "{%4, %5, %6, %7}, \t"
                    "{%8, %9}, \t"
                    "{%0, %1, %2, %3}; ":
                    "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                    "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                    "r"(a_fragment_int[0 + 8 * i]), "r"(a_fragment_int[4 + 8 * i]),
                    "r"(a_fragment_int[1 + 8 * i]), "r"(a_fragment_int[5 + 8 * i]),
                    "r"(b_fragment_int[0 + 4 * j]), "r"(b_fragment_int[1 + 4 * j])
                );
                __syncthreads();
                asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 \t"
                    "{%0, %1, %2, %3}, \t"
                    "{%4, %5, %6, %7}, \t"
                    "{%8, %9}, \t"
                    "{%0, %1, %2, %3}; ":
                    "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                    "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                    "r"(a_fragment_int[2 + 8 * i]), "r"(a_fragment_int[6 + 8 * i]),
                    "r"(a_fragment_int[3 + 8 * i]), "r"(a_fragment_int[7 + 8 * i]),
                    "r"(b_fragment_int[2 + 4 * j]), "r"(b_fragment_int[3 + 4 * j])
                );
            }
        }// end mma compute
        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);
        __syncthreads();

    }// end main loop
    int sel_com = ((K / Block_K) & 1) ^ 1;
    float4 * smem_a_sel = reinterpret_cast<float4 *>(smem_a + sel_com * smem_a_offset + warp_x * (smem_a_offset >> 1));
    float4 * smem_b_sel = reinterpret_cast<float4 *>(smem_b + sel_com * smem_b_offset + warp_y * (smem_b_offset >> 1));
    matrix_a_fragment[0] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * (4+0));
    matrix_a_fragment[1] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * (4+0));
    matrix_a_fragment[2] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 16) * (4+0));
    matrix_a_fragment[3] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 24) * (4+0));
    matrix_a_fragment[4] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 32) * (4+0));
    matrix_a_fragment[5] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 40) * (4+0));
    matrix_a_fragment[6] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 48) * (4+0));
    matrix_a_fragment[7] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 56) * (4+0));
    matrix_b_fragment[0] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * (4+0));
    matrix_b_fragment[1] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * (4+0));
    matrix_b_fragment[2] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 16) * (4+0));
    matrix_b_fragment[3] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 24) * (4+0));
    matrix_b_fragment[4] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 32) * (4+0));
    matrix_b_fragment[5] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 40) * (4+0));
    matrix_b_fragment[6] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 48) * (4+0));
    matrix_b_fragment[7] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 56) * (4+0));

    int * a_fragment_int = reinterpret_cast<int *>(matrix_a_fragment);// 8 float4 --> 32 reg
    int * b_fragment_int = reinterpret_cast<int *>(matrix_b_fragment);// 8 float4 --> 32 reg

    #pragma unroll
    for(int i = 0; i < 4; i++){
        #pragma unroll
        for(int j = 0; j < 8; j++){
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[0 + 8 * i]), "r"(a_fragment_int[4 + 8 * i]),
                "r"(a_fragment_int[1 + 8 * i]), "r"(a_fragment_int[5 + 8 * i]),
                "r"(b_fragment_int[0 + 4 * j]), "r"(b_fragment_int[1 + 4 * j])
            );
            __syncthreads();
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[2 + 8 * i]), "r"(a_fragment_int[6 + 8 * i]),
                "r"(a_fragment_int[3 + 8 * i]), "r"(a_fragment_int[7 + 8 * i]),
                "r"(b_fragment_int[2 + 4 * j]), "r"(b_fragment_int[3 + 4 * j])
            );
        }
    }// end mma compute
    // __syncthreads();
    int store_gmem_m = bx * Block_M + warp_x * 64;
    int store_gmem_n = by * Block_N + warp_y * 64;
    float2 * output_ = reinterpret_cast<float2 *>(Output_Value + (store_gmem_m + (int)(lane_id / 4)) * N + store_gmem_n + ((lane_id % 4) * 2));
    float2 * output_fragment_ = reinterpret_cast<float2 *>(output_fragment);// 128 reg --> 64 float2
    #pragma unroll
    for(int i = 0; i < 4; i++){
        #pragma unroll
        for(int j = 0; j < 8; j++){
            *(output_ + (int)(( (i*16  ) * N + j * 8 )/2)) = *(output_fragment_ + 2 * (i*8+j)    );
            *(output_ + (int)(( (i*16+8) * N + j * 8 )/2)) = *(output_fragment_ + 2 * (i*8+j) + 1);
        }
    }
}
hipError_t GEMMex(
    int M, int K, int N,
    const e4m3 * __restrict__ A_Value,
    const e4m3 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    const int Block_M = 128, Block_N = 128, Block_K = 64;
    dim3 block_dim(128,1,1);
	dim3 grid_dim(ceil(static_cast<float>(M) / Block_M), ceil(static_cast<float>(N) / Block_N), 1);

    GEMM_e4m3_e4m3_o32_stage2_row_col<<<grid_dim, block_dim>>>(
        M, K, N, A_Value, B_Value, Output_Value);

	return hipGetLastError();
}
hipError_t GEMM(
    int M, int K, int N,
    const e4m3 * __restrict__ A_Value,
    const e4m3 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
	return GEMMex(M, K, N, A_Value, B_Value, Output_Value);
}
//kernel2////////////////////e4m3 * e4m3 = f32//////////////////
__global__ void GEMM_e4m3_e4m3_o32_stage4_row_col(
    int M, int K, int N,
    const e4m3 * __restrict__ A_Value,
    const e4m3 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    // thread num = 128
    constexpr int Block_M = 128;
    constexpr int Block_K = 64;
    constexpr int Block_N = 128;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int lane_id = tid % 32;
    int wid = tid / 32;// 128 / 32 = 4

    if(bx >= M / Block_M || by >= N / Block_N){
        return;
    }
    constexpr int APAD = 0;
    constexpr int BPAD = 0;


    constexpr int smem_a_offset = Block_M * (Block_K + APAD);
    constexpr int smem_b_offset = Block_N * (Block_K + BPAD);
    // const int smem_size_float4 = 4 * (smem_a_offset + smem_b_offset) / 16;
    // __shared__ float4 smem[smem_size_float4];
    __shared__ e4m3 smem_a[4 * smem_a_offset];
    __shared__ e4m3 smem_b[4 * smem_b_offset];
    // e4m3 * smem_a = smem;
    // e4m3 * smem_b = smem_a + 4 * smem_a_offset;

    // __align__(32) float4 matrix_a_fragment[8];// 8 float4 = 32 reg
    // __align__(32) float4 matrix_b_fragment[8];// 8 float4 = 32 reg
    // __align__(32) float output_fragment[128];// 4(m) * 8(n) * 4(one tile) = 128 float
    float4 matrix_a_fragment[8];// 8 float4 = 32 reg
    float4 matrix_b_fragment[8];// 8 float4 = 32 reg
    float output_fragment[128];// 4(m) * 8(n) * 4(one tile) = 128 float

    int smem_a_base_addr = __cvta_generic_to_shared(smem_a);
    int smem_b_base_addr = __cvta_generic_to_shared(smem_b);
    int smem_a_m = (tid / 4) * 4;
    int smem_a_k = (tid % 4) * 16;
    int smem_b_k = (tid % 4) * 16;
    int smem_b_n = (tid / 4) * 4;
    // block_size = 128 * 64 
    // thred num = 128 
    // each thread load = 128 * 64 / 128 = 64 fp8 = 4 float4
    int smem_a_addr0 = smem_a_base_addr + (smem_a_m * (Block_K + APAD) + smem_a_k) * sizeof(char);
    int smem_a_addr1 = smem_a_addr0 + 1 * (Block_K + APAD) * sizeof(char);
    int smem_a_addr2 = smem_a_addr0 + 2 * (Block_K + APAD) * sizeof(char);
    int smem_a_addr3 = smem_a_addr0 + 3 * (Block_K + APAD) * sizeof(char);
    
    int smem_b_addr0 = smem_b_base_addr + (smem_b_n * (Block_K + BPAD) + smem_b_k) * sizeof(char);
    int smem_b_addr1 = smem_b_addr0 + 1 * (Block_K + BPAD) * sizeof(char);
    int smem_b_addr2 = smem_b_addr0 + 2 * (Block_K + BPAD) * sizeof(char);
    int smem_b_addr3 = smem_b_addr0 + 3 * (Block_K + BPAD) * sizeof(char);

    int gmem_a_m = bx * Block_M + smem_a_m;
    int gmem_a_k = smem_a_k;
    int gmem_b_k = smem_b_k;
    int gmem_b_n = by * Block_N + smem_b_n;

    int gmem_a_addr = gmem_a_m * K + gmem_a_k;
    int gmem_b_addr = gmem_b_n * K + gmem_b_k;
    // stage 1
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr0), "l"(&A_Value[gmem_a_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr1), "l"(&A_Value[gmem_a_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr2), "l"(&A_Value[gmem_a_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr3), "l"(&A_Value[gmem_a_addr + 3 * K]));

    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr0), "l"(&B_Value[gmem_b_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr1), "l"(&B_Value[gmem_b_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr2), "l"(&B_Value[gmem_b_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr3), "l"(&B_Value[gmem_b_addr + 3 * K]));
    // stage 2
    gmem_a_addr += Block_K;
    gmem_b_addr += Block_K;
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr0+smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr1+smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr2+smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr3+smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 3 * K]));

    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr0+smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr1+smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr2+smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr3+smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 3 * K]));

    // stage 3
    gmem_a_addr += Block_K;
    gmem_b_addr += Block_K;
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr0+2*smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr1+2*smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr2+2*smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr3+2*smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 3 * K]));

    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr0+2*smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr1+2*smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr2+2*smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr3+2*smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 3 * K]));

    asm ("cp.async.commit_group;\n" ::);
    asm ("cp.async.wait_group 0;\n" ::);
    __syncthreads();

    int warp_x = wid % 2;//0 1 0 1
    int warp_y = wid / 2;//0 0 1 1
    #pragma unroll 32
    for(int bk = 3; bk < K/Block_K; bk++){
        int sel_mem = bk % 4; // 3 0 1 2 3 0 1
        int sel_com = (bk - 3) % 4; // 0 1 2 3 0 1 2 3
        gmem_a_addr += Block_K;
        gmem_b_addr += Block_K;
        
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr0 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr1 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 1 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr2 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 2 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr3 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 3 * K]));

        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr0 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr1 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 1 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr2 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 2 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr3 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 3 * K]));
        
        float4 * smem_a_sel = reinterpret_cast<float4 *>(smem_a + sel_com * smem_a_offset + warp_x * 64 * (Block_K+APAD));
        float4 * smem_b_sel = reinterpret_cast<float4 *>(smem_b + sel_com * smem_b_offset + warp_y * 64 * (Block_K+BPAD));
        matrix_a_fragment[0] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
        matrix_a_fragment[1] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
        matrix_a_fragment[2] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
        matrix_a_fragment[3] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
        matrix_a_fragment[4] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
        matrix_a_fragment[5] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
        matrix_a_fragment[6] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
        matrix_a_fragment[7] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);
        matrix_b_fragment[0] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
        matrix_b_fragment[1] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
        matrix_b_fragment[2] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
        matrix_b_fragment[3] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
        matrix_b_fragment[4] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
        matrix_b_fragment[5] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
        matrix_b_fragment[6] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
        matrix_b_fragment[7] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);

        int * a_fragment_int = reinterpret_cast<int *>(matrix_a_fragment);// 8 float4 --> 32 reg
        int * b_fragment_int = reinterpret_cast<int *>(matrix_b_fragment);// 8 float4 --> 32 reg

        #pragma unroll
        for(int i = 0; i < 4; i++){
            #pragma unroll
            for(int j = 0; j < 8; j++){
                asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 \t"
                    "{%0, %1, %2, %3}, \t"
                    "{%4, %5, %6, %7}, \t"
                    "{%8, %9}, \t"
                    "{%0, %1, %2, %3}; ":
                    "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                    "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                    "r"(a_fragment_int[0 + 8 * i]), "r"(a_fragment_int[4 + 8 * i]),
                    "r"(a_fragment_int[1 + 8 * i]), "r"(a_fragment_int[5 + 8 * i]),
                    "r"(b_fragment_int[0 + 4 * j]), "r"(b_fragment_int[1 + 4 * j])
                );
                // __syncthreads();
                asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 \t"
                    "{%0, %1, %2, %3}, \t"
                    "{%4, %5, %6, %7}, \t"
                    "{%8, %9}, \t"
                    "{%0, %1, %2, %3}; ":
                    "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                    "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                    "r"(a_fragment_int[2 + 8 * i]), "r"(a_fragment_int[6 + 8 * i]),
                    "r"(a_fragment_int[3 + 8 * i]), "r"(a_fragment_int[7 + 8 * i]),
                    "r"(b_fragment_int[2 + 4 * j]), "r"(b_fragment_int[3 + 4 * j])
                );
            }
        }// end mma compute
        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);
        __syncthreads();

    }// end main loop
    int sel_com = ((int)(K / 64) - 3) % 4;
    float4 * smem_a_sel = reinterpret_cast<float4 *>(smem_a + sel_com * smem_a_offset + warp_x * 64 * (Block_K+APAD));
    float4 * smem_b_sel = reinterpret_cast<float4 *>(smem_b + sel_com * smem_b_offset + warp_y * 64 * (Block_K+BPAD));
    matrix_a_fragment[0] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
    matrix_a_fragment[1] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
    matrix_a_fragment[2] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
    matrix_a_fragment[3] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
    matrix_a_fragment[4] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
    matrix_a_fragment[5] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
    matrix_a_fragment[6] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
    matrix_a_fragment[7] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);
    matrix_b_fragment[0] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
    matrix_b_fragment[1] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
    matrix_b_fragment[2] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
    matrix_b_fragment[3] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
    matrix_b_fragment[4] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
    matrix_b_fragment[5] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
    matrix_b_fragment[6] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
    matrix_b_fragment[7] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);

    int * a_fragment_int = reinterpret_cast<int *>(matrix_a_fragment);// 8 float4 --> 32 reg
    int * b_fragment_int = reinterpret_cast<int *>(matrix_b_fragment);// 8 float4 --> 32 reg

    #pragma unroll
    for(int i = 0; i < 4; i++){
        #pragma unroll
        for(int j = 0; j < 8; j++){
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[0 + 8 * i]), "r"(a_fragment_int[4 + 8 * i]),
                "r"(a_fragment_int[1 + 8 * i]), "r"(a_fragment_int[5 + 8 * i]),
                "r"(b_fragment_int[0 + 4 * j]), "r"(b_fragment_int[1 + 4 * j])
            );
            // __syncthreads();
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[2 + 8 * i]), "r"(a_fragment_int[6 + 8 * i]),
                "r"(a_fragment_int[3 + 8 * i]), "r"(a_fragment_int[7 + 8 * i]),
                "r"(b_fragment_int[2 + 4 * j]), "r"(b_fragment_int[3 + 4 * j])
            );
        }
    }// end mma compute
    __syncthreads();
    sel_com = ((int)(K / 64)-2) % 4;
    smem_a_sel = reinterpret_cast<float4 *>(smem_a + sel_com * smem_a_offset + warp_x * 64 * (Block_K+APAD));
    smem_b_sel = reinterpret_cast<float4 *>(smem_b + sel_com * smem_b_offset + warp_y * 64 * (Block_K+BPAD));
    matrix_a_fragment[0] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
    matrix_a_fragment[1] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
    matrix_a_fragment[2] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
    matrix_a_fragment[3] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
    matrix_a_fragment[4] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
    matrix_a_fragment[5] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
    matrix_a_fragment[6] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
    matrix_a_fragment[7] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);
    matrix_b_fragment[0] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
    matrix_b_fragment[1] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
    matrix_b_fragment[2] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
    matrix_b_fragment[3] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
    matrix_b_fragment[4] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
    matrix_b_fragment[5] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
    matrix_b_fragment[6] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
    matrix_b_fragment[7] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);

    a_fragment_int = reinterpret_cast<int *>(matrix_a_fragment);// 8 float4 --> 32 reg
    b_fragment_int = reinterpret_cast<int *>(matrix_b_fragment);// 8 float4 --> 32 reg

    #pragma unroll
    for(int i = 0; i < 4; i++){
        #pragma unroll
        for(int j = 0; j < 8; j++){
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[0 + 8 * i]), "r"(a_fragment_int[4 + 8 * i]),
                "r"(a_fragment_int[1 + 8 * i]), "r"(a_fragment_int[5 + 8 * i]),
                "r"(b_fragment_int[0 + 4 * j]), "r"(b_fragment_int[1 + 4 * j])
            );
            // __syncthreads();
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[2 + 8 * i]), "r"(a_fragment_int[6 + 8 * i]),
                "r"(a_fragment_int[3 + 8 * i]), "r"(a_fragment_int[7 + 8 * i]),
                "r"(b_fragment_int[2 + 4 * j]), "r"(b_fragment_int[3 + 4 * j])
            );
        }
    }// end mma compute
    __syncthreads();
    sel_com = ((int)(K / 64)-1) % 4;
    smem_a_sel = reinterpret_cast<float4 *>(smem_a + sel_com * smem_a_offset + warp_x * 64 * (Block_K+APAD));
    smem_b_sel = reinterpret_cast<float4 *>(smem_b + sel_com * smem_b_offset + warp_y * 64 * (Block_K+BPAD));
    matrix_a_fragment[0] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
    matrix_a_fragment[1] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
    matrix_a_fragment[2] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
    matrix_a_fragment[3] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
    matrix_a_fragment[4] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
    matrix_a_fragment[5] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
    matrix_a_fragment[6] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
    matrix_a_fragment[7] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);
    matrix_b_fragment[0] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
    matrix_b_fragment[1] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
    matrix_b_fragment[2] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
    matrix_b_fragment[3] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
    matrix_b_fragment[4] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
    matrix_b_fragment[5] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
    matrix_b_fragment[6] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
    matrix_b_fragment[7] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);

    a_fragment_int = reinterpret_cast<int *>(matrix_a_fragment);// 8 float4 --> 32 reg
    b_fragment_int = reinterpret_cast<int *>(matrix_b_fragment);// 8 float4 --> 32 reg

    #pragma unroll
    for(int i = 0; i < 4; i++){
        #pragma unroll
        for(int j = 0; j < 8; j++){
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[0 + 8 * i]), "r"(a_fragment_int[4 + 8 * i]),
                "r"(a_fragment_int[1 + 8 * i]), "r"(a_fragment_int[5 + 8 * i]),
                "r"(b_fragment_int[0 + 4 * j]), "r"(b_fragment_int[1 + 4 * j])
            );
            // __syncthreads();
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e4m3.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[2 + 8 * i]), "r"(a_fragment_int[6 + 8 * i]),
                "r"(a_fragment_int[3 + 8 * i]), "r"(a_fragment_int[7 + 8 * i]),
                "r"(b_fragment_int[2 + 4 * j]), "r"(b_fragment_int[3 + 4 * j])
            );
        }
    }// end mma compute
    __syncthreads();

    float2 * output_ = reinterpret_cast<float2 *>(Output_Value + (bx * Block_M + warp_x * 64 + (int)(lane_id / 4)) * N + by * Block_N + warp_y * 64 + ((lane_id % 4) * 2));
    float2 * output_fragment_ = reinterpret_cast<float2 *>(output_fragment);// 128 reg --> 64 float2
    #pragma unroll
    for(int i = 0; i < 4; i++){
        #pragma unroll
        for(int j = 0; j < 8; j++){
            *(output_ + (int)(( (i*16  ) * N + j * 8 )/2)) = *(output_fragment_ + 2 * (i*8+j)    );
            *(output_ + (int)(( (i*16+8) * N + j * 8 )/2)) = *(output_fragment_ + 2 * (i*8+j) + 1);
        }
    }
}
hipError_t GEMMex4(
    int M, int K, int N,
    const e4m3 * __restrict__ A_Value,
    const e4m3 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    const int Block_M = 128, Block_N = 128, Block_K = 64;
    dim3 block_dim(128,1,1);
	dim3 grid_dim(ceil(static_cast<float>(M) / Block_M), ceil(static_cast<float>(N) / Block_N), 1);
    // unsigned int dsmem = 4 * (Block_M * (Block_K + 0) + Block_N * (Block_K + 0)) * sizeof(char);
    // cudaFuncSetAttribute(GEMM_e4m3_e4m3_o32_stage4_row_col,
        // cudaFuncAttributeMaxDynamicSharedMemorySize, 233472);

    GEMM_e4m3_e4m3_o32_stage4_row_col<<<grid_dim, block_dim>>>(
        M, K, N, A_Value, B_Value, Output_Value);
	return hipGetLastError();
}
hipError_t GEMM4(
    int M, int K, int N,
    const e4m3 * __restrict__ A_Value,
    const e4m3 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
	return GEMMex(M, K, N, A_Value, B_Value, Output_Value);
}
//kernel3////////////////////e5m2 * e5m2 = f32//////////////////
__global__ void GEMM_e5m2_e5m2_o32_stage2_row_col(
    int M, int K, int N,
    const e5m2 * __restrict__ A_Value,
    const e5m2 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    // thread num = 128
    constexpr int Block_M = 128;
    constexpr int Block_K = 64;
    constexpr int Block_N = 128;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int lane_id = threadIdx.x % 32;
    int wid = tid / 32;// 128 / 32 = 4

    if(bx >= M / Block_M || by >= N / Block_N){
        return;
    }
    const int APAD = 0;
    const int BPAD = 0;

    // extern __shared__ float4 smem[];
    const int smem_a_offset = Block_M * (Block_K + APAD);
    const int smem_b_offset = Block_N * (Block_K + BPAD);
    // const int smem_size_float4 = 2 * (smem_a_offset + smem_b_offset) / 16;
    __shared__ e5m2 smem_a[smem_a_offset * 2];
    __shared__ e5m2 smem_b[smem_b_offset * 2];
    // e5m2 * smem_a = reinterpret_cast<e5m2 *>(smem);
    // e5m2 * smem_b = smem_a + 2 * smem_a_offset;
    float4 matrix_a_fragment[8];// 8 float4 = 32 reg
    float4 matrix_b_fragment[8];// 8 float4 = 32 reg
    float output_fragment[128];// 4(m) * 8(n) * 4(one tile) = 128 float

    int smem_a_m = (int)(tid / 4) * 4;
    int smem_a_k = (tid % 4) * 16;
    int smem_b_k = (tid % 4) * 16;
    int smem_b_n = (int)(tid / 4) * 4;
    int smem_a_base_addr = __cvta_generic_to_shared(smem_a);
    int smem_b_base_addr = __cvta_generic_to_shared(smem_b);

    int smem_a_addr0 = smem_a_base_addr + (smem_a_m * (Block_K + APAD) + smem_a_k) * sizeof(char);
    int smem_a_addr1 = smem_a_addr0 + 1 * (Block_K + APAD) * sizeof(char);
    int smem_a_addr2 = smem_a_addr0 + 2 * (Block_K + APAD) * sizeof(char);
    int smem_a_addr3 = smem_a_addr0 + 3 * (Block_K + APAD) * sizeof(char);
    
    int smem_b_addr0 = smem_b_base_addr + (smem_b_n * (Block_K + BPAD) + smem_b_k) * sizeof(char);
    int smem_b_addr1 = smem_b_addr0 + 1 * (Block_K + BPAD) * sizeof(char);
    int smem_b_addr2 = smem_b_addr0 + 2 * (Block_K + BPAD) * sizeof(char);
    int smem_b_addr3 = smem_b_addr0 + 3 * (Block_K + BPAD) * sizeof(char);

    int gmem_a_m = bx * Block_M + smem_a_m;
    int gmem_a_k = smem_a_k;
    int gmem_b_k = smem_b_k;
    int gmem_b_n = by * Block_N + smem_b_n;

    int gmem_a_addr = gmem_a_m * K + gmem_a_k;
    int gmem_b_addr = gmem_b_n * K + gmem_b_k;

    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr0), "l"(&A_Value[gmem_a_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr1), "l"(&A_Value[gmem_a_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr2), "l"(&A_Value[gmem_a_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr3), "l"(&A_Value[gmem_a_addr + 3 * K]));

    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr0), "l"(&B_Value[gmem_b_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr1), "l"(&B_Value[gmem_b_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr2), "l"(&B_Value[gmem_b_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr3), "l"(&B_Value[gmem_b_addr + 3 * K]));

    asm ("cp.async.commit_group;\n" ::);
    asm ("cp.async.wait_group 0;\n" ::);
    __syncthreads();

    int warp_x = wid % 2;//0 1 0 1
    int warp_y = wid / 2;//0 0 1 1
    #pragma unroll 16
    for(int bk = 1; bk < K/Block_K; bk++){
        int sel = (bk & 1) ^ 1; // 0 1 0 1
        int sel_mem = ((bk - 1) & 1) ^ 1; // 1 0 1 0
        gmem_a_addr += Block_K;
        gmem_b_addr += Block_K;
        
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr0 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr1 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 1 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr2 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 2 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr3 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 3 * K]));

        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr0 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr1 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 1 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr2 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 2 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr3 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 3 * K]));
        
        float4 * smem_a_sel = reinterpret_cast<float4 *>(smem_a + sel * smem_a_offset + warp_x * (int)(smem_a_offset/2));
        float4 * smem_b_sel = reinterpret_cast<float4 *>(smem_b + sel * smem_b_offset + warp_y * (int)(smem_b_offset/2));
        matrix_a_fragment[0] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
        matrix_a_fragment[1] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
        matrix_a_fragment[2] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
        matrix_a_fragment[3] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
        matrix_a_fragment[4] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
        matrix_a_fragment[5] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
        matrix_a_fragment[6] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
        matrix_a_fragment[7] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);
        matrix_b_fragment[0] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
        matrix_b_fragment[1] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
        matrix_b_fragment[2] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
        matrix_b_fragment[3] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
        matrix_b_fragment[4] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
        matrix_b_fragment[5] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
        matrix_b_fragment[6] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
        matrix_b_fragment[7] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);

        int * a_fragment_int = reinterpret_cast<int *>(matrix_a_fragment);// 8 float4 --> 32 reg
        int * b_fragment_int = reinterpret_cast<int *>(matrix_b_fragment);// 8 float4 --> 32 reg

        #pragma unroll
        for(int i = 0; i < 4; i++){
            #pragma unroll
            for(int j = 0; j < 8; j++){
                asm ("mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 \t"
                    "{%0, %1, %2, %3}, \t"
                    "{%4, %5, %6, %7}, \t"
                    "{%8, %9}, \t"
                    "{%0, %1, %2, %3}; ":
                    "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                    "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                    "r"(a_fragment_int[0 + 8 * i]), "r"(a_fragment_int[4 + 8 * i]),
                    "r"(a_fragment_int[1 + 8 * i]), "r"(a_fragment_int[5 + 8 * i]),
                    "r"(b_fragment_int[0 + 4 * j]), "r"(b_fragment_int[1 + 4 * j])
                );
                asm ("mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 \t"
                    "{%0, %1, %2, %3}, \t"
                    "{%4, %5, %6, %7}, \t"
                    "{%8, %9}, \t"
                    "{%0, %1, %2, %3}; ":
                    "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                    "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                    "r"(a_fragment_int[2 + 8 * i]), "r"(a_fragment_int[6 + 8 * i]),
                    "r"(a_fragment_int[3 + 8 * i]), "r"(a_fragment_int[7 + 8 * i]),
                    "r"(b_fragment_int[2 + 4 * j]), "r"(b_fragment_int[3 + 4 * j])
                );
            }
        }// end mma compute
        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);
        __syncthreads();

    }// end main loop
    int sel = ((K / Block_K) & 1) ^ 1;
    float4 * smem_a_sel = reinterpret_cast<float4 *>(smem_a + sel * smem_a_offset + warp_x * (int)(smem_a_offset/2));
    float4 * smem_b_sel = reinterpret_cast<float4 *>(smem_b + sel * smem_b_offset + warp_y * (int)(smem_b_offset/2));
    matrix_a_fragment[0] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
    matrix_a_fragment[1] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
    matrix_a_fragment[2] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
    matrix_a_fragment[3] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
    matrix_a_fragment[4] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
    matrix_a_fragment[5] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
    matrix_a_fragment[6] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
    matrix_a_fragment[7] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);
    matrix_b_fragment[0] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
    matrix_b_fragment[1] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
    matrix_b_fragment[2] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
    matrix_b_fragment[3] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
    matrix_b_fragment[4] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
    matrix_b_fragment[5] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
    matrix_b_fragment[6] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
    matrix_b_fragment[7] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);

    int * a_fragment_int = reinterpret_cast<int *>(matrix_a_fragment);// 8 float4 --> 32 reg
    int * b_fragment_int = reinterpret_cast<int *>(matrix_b_fragment);// 8 float4 --> 32 reg

    #pragma unroll
    for(int i = 0; i < 4; i++){
        #pragma unroll
        for(int j = 0; j < 8; j++){
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[0 + 8 * i]), "r"(a_fragment_int[4 + 8 * i]),
                "r"(a_fragment_int[1 + 8 * i]), "r"(a_fragment_int[5 + 8 * i]),
                "r"(b_fragment_int[0 + 4 * j]), "r"(b_fragment_int[1 + 4 * j])
            );
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e5m2.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[2 + 8 * i]), "r"(a_fragment_int[6 + 8 * i]),
                "r"(a_fragment_int[3 + 8 * i]), "r"(a_fragment_int[7 + 8 * i]),
                "r"(b_fragment_int[2 + 4 * j]), "r"(b_fragment_int[3 + 4 * j])
            );
        }
    }// end mma compute
    __syncthreads();
    float2 * output_ = reinterpret_cast<float2 *>(Output_Value + (bx * Block_M + warp_x * 64 + (int)(lane_id / 4)) * N + by * Block_N + warp_y * 64 + (int)((lane_id % 4) * 2));
    float2 * output_fragment_ = reinterpret_cast<float2 *>(output_fragment);// 128 reg --> 64 float2
    #pragma unroll
    for(int i = 0; i < 4; i++){
        #pragma unroll
        for(int j = 0; j < 8; j++){
            *(output_ + (i*8  ) * N + j * 4) = *(output_fragment_ + 2 * (i*8+j)    );
            *(output_ + (i*8+4) * N + j * 4) = *(output_fragment_ + 2 * (i*8+j) + 1);
        }
    }
}

hipError_t GEMMex(
    int M, int K, int N,
    const e5m2 * __restrict__ A_Value,
    const e5m2 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    const int Block_M = 128, Block_N = 128, Block_K = 64;
    dim3 block_dim(128,1,1);
	dim3 grid_dim(ceil(static_cast<float>(M) / Block_M), ceil(static_cast<float>(N) / Block_N), 1);

    GEMM_e5m2_e5m2_o32_stage2_row_col<<<grid_dim, block_dim>>>(
        M, K, N, A_Value, B_Value, Output_Value);
	return hipGetLastError();
}
hipError_t GEMM(
    int M, int K, int N,
    const e5m2 * __restrict__ A_Value,
    const e5m2 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
	return GEMMex(M, K, N, A_Value, B_Value, Output_Value);
}
//kernel5////////////////////e4m3 * e5m2 = f32//////////////////
__global__ void GEMM_e4m3_e5m2_o32_stage2_row_col(
    int M, int K, int N,
    const e4m3 * __restrict__ A_Value,
    const e5m2 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    // thread num = 128
    constexpr int Block_M = 128;
    constexpr int Block_K = 64;
    constexpr int Block_N = 128;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int lane_id = threadIdx.x % 32;
    int wid = tid >> 5;// 128 / 32 = 4

    if(bx >= M / Block_M || by >= N / Block_N){
        return;
    }
    constexpr int APAD = 0;
    constexpr int BPAD = 0;

    // extern __shared__ float4 smem[];
    constexpr int smem_a_offset = Block_M * (Block_K + APAD);
    constexpr int smem_b_offset = Block_N * (Block_K + BPAD);
    const int smem_size_float4 = 2 * (smem_a_offset + smem_b_offset) / 16;
    __shared__ float4 smem[smem_size_float4];
    e4m3 * smem_a = reinterpret_cast<e4m3 *>(smem);
    e5m2 * smem_b = reinterpret_cast<e5m2 *>(smem) + 2 * smem_a_offset;

    // __align__(32) float4 matrix_a_fragment[8];// 8 float4 = 32 reg
    // __align__(32) float4 matrix_b_fragment[8];// 8 float4 = 32 reg
    // __align__(32) float output_fragment[128];// 4(m) * 8(n) * 4(one tile) = 128 float
    float4 matrix_a_fragment[8];// 8 float4 = 32 reg
    float4 matrix_b_fragment[8];// 8 float4 = 32 reg
    float output_fragment[128];// 4(m) * 8(n) * 4(one tile) = 128 float

    int smem_a_base_addr = __cvta_generic_to_shared(smem_a);
    int smem_b_base_addr = __cvta_generic_to_shared(smem_b);
    int smem_a_m = (tid / 4) << 2;
    int smem_a_k = (tid &  3) << 4;
    int smem_b_k = (tid &  3) << 4;
    int smem_b_n = (tid / 4) << 2;
    // block_size = 128 * 64 
    // thred num = 128 
    // each thread load = 128 * 64 / 128 = 64 fp8 = 4 float4
    int smem_a_addr0 = smem_a_base_addr + (smem_a_m * (Block_K + APAD) + smem_a_k) * sizeof(char);
    int smem_a_addr1 = smem_a_addr0 + 1 * (Block_K + APAD) * sizeof(char);
    int smem_a_addr2 = smem_a_addr0 + 2 * (Block_K + APAD) * sizeof(char);
    int smem_a_addr3 = smem_a_addr0 + 3 * (Block_K + APAD) * sizeof(char);
    
    int smem_b_addr0 = smem_b_base_addr + (smem_b_n * (Block_K + BPAD) + smem_b_k) * sizeof(char);
    int smem_b_addr1 = smem_b_addr0 + 1 * (Block_K + BPAD) * sizeof(char);
    int smem_b_addr2 = smem_b_addr0 + 2 * (Block_K + BPAD) * sizeof(char);
    int smem_b_addr3 = smem_b_addr0 + 3 * (Block_K + BPAD) * sizeof(char);

    int gmem_a_m = bx * Block_M + smem_a_m;
    int gmem_a_k = smem_a_k;
    int gmem_b_k = smem_b_k;
    int gmem_b_n = by * Block_N + smem_b_n;

    int gmem_a_addr = gmem_a_m * K + gmem_a_k;
    int gmem_b_addr = gmem_b_n * K + gmem_b_k;

    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr0), "l"(&A_Value[gmem_a_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr1), "l"(&A_Value[gmem_a_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr2), "l"(&A_Value[gmem_a_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr3), "l"(&A_Value[gmem_a_addr + 3 * K]));

    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr0), "l"(&B_Value[gmem_b_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr1), "l"(&B_Value[gmem_b_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr2), "l"(&B_Value[gmem_b_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr3), "l"(&B_Value[gmem_b_addr + 3 * K]));

    asm ("cp.async.commit_group;\n" ::);
    asm ("cp.async.wait_group 0;\n" ::);
    __syncthreads();

    int warp_x = wid &  1;//0 1 0 1
    int warp_y = wid >> 1;//0 0 1 1
    #pragma unroll 16
    for(int bk = 1; bk < K/Block_K; bk++){
        int sel = (bk & 1) ^ 1; // 0 1 0 1
        int sel_mem = ((bk - 1) & 1) ^ 1; // 1 0 1 0
        gmem_a_addr += Block_K;
        gmem_b_addr += Block_K;
        
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr0 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr1 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 1 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr2 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 2 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr3 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 3 * K]));

        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr0 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr1 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 1 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr2 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 2 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr3 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 3 * K]));
        
        float4 * smem_a_sel = reinterpret_cast<float4 *>(smem_a + sel * smem_a_offset + warp_x * (smem_a_offset >> 1));
        float4 * smem_b_sel = reinterpret_cast<float4 *>(smem_b + sel * smem_b_offset + warp_y * (smem_b_offset >> 1));
        matrix_a_fragment[0] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
        matrix_a_fragment[1] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
        matrix_a_fragment[2] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
        matrix_a_fragment[3] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
        matrix_a_fragment[4] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
        matrix_a_fragment[5] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
        matrix_a_fragment[6] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
        matrix_a_fragment[7] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);
        matrix_b_fragment[0] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
        matrix_b_fragment[1] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
        matrix_b_fragment[2] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
        matrix_b_fragment[3] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
        matrix_b_fragment[4] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
        matrix_b_fragment[5] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
        matrix_b_fragment[6] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
        matrix_b_fragment[7] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);

        int * a_fragment_int = reinterpret_cast<int *>(matrix_a_fragment);// 8 float4 --> 32 reg
        int * b_fragment_int = reinterpret_cast<int *>(matrix_b_fragment);// 8 float4 --> 32 reg

        #pragma unroll
        for(int i = 0; i < 4; i++){
            #pragma unroll
            for(int j = 0; j < 8; j++){
                asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e5m2.f32 \t"
                    "{%0, %1, %2, %3}, \t"
                    "{%4, %5, %6, %7}, \t"
                    "{%8, %9}, \t"
                    "{%0, %1, %2, %3}; ":
                    "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                    "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                    "r"(a_fragment_int[0 + 8 * i]), "r"(a_fragment_int[1 + 8 * i]),
                    "r"(a_fragment_int[4 + 8 * i]), "r"(a_fragment_int[5 + 8 * i]),
                    "r"(b_fragment_int[0 + 4 * j]), "r"(b_fragment_int[1 + 4 * j])
                );
                asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e5m2.f32 \t"
                    "{%0, %1, %2, %3}, \t"
                    "{%4, %5, %6, %7}, \t"
                    "{%8, %9}, \t"
                    "{%0, %1, %2, %3}; ":
                    "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                    "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                    "r"(a_fragment_int[2 + 8 * i]), "r"(a_fragment_int[3 + 8 * i]),
                    "r"(a_fragment_int[6 + 8 * i]), "r"(a_fragment_int[7 + 8 * i]),
                    "r"(b_fragment_int[2 + 4 * j]), "r"(b_fragment_int[3 + 4 * j])
                );
            }
        }// end mma compute
        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);
        __syncthreads();

    }// end main loop
    int sel = ((K / Block_K) & 1) ^ 1;
    float4 * smem_a_sel = reinterpret_cast<float4 *>(smem_a + sel * smem_a_offset + warp_x * (smem_a_offset >> 1));
    float4 * smem_b_sel = reinterpret_cast<float4 *>(smem_b + sel * smem_b_offset + warp_y * (smem_b_offset >> 1));
    matrix_a_fragment[0] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
    matrix_a_fragment[1] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
    matrix_a_fragment[2] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
    matrix_a_fragment[3] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
    matrix_a_fragment[4] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
    matrix_a_fragment[5] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
    matrix_a_fragment[6] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
    matrix_a_fragment[7] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);
    matrix_b_fragment[0] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
    matrix_b_fragment[1] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
    matrix_b_fragment[2] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
    matrix_b_fragment[3] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
    matrix_b_fragment[4] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
    matrix_b_fragment[5] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
    matrix_b_fragment[6] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
    matrix_b_fragment[7] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);

    int * a_fragment_int = reinterpret_cast<int *>(matrix_a_fragment);// 8 float4 --> 32 reg
    int * b_fragment_int = reinterpret_cast<int *>(matrix_b_fragment);// 8 float4 --> 32 reg

    #pragma unroll
    for(int i = 0; i < 4; i++){
        #pragma unroll
        for(int j = 0; j < 8; j++){
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e5m2.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[0 + 8 * i]), "r"(a_fragment_int[1 + 8 * i]),
                "r"(a_fragment_int[4 + 8 * i]), "r"(a_fragment_int[5 + 8 * i]),
                "r"(b_fragment_int[0 + 4 * j]), "r"(b_fragment_int[1 + 4 * j])
            );
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e4m3.e5m2.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[2 + 8 * i]), "r"(a_fragment_int[3 + 8 * i]),
                "r"(a_fragment_int[6 + 8 * i]), "r"(a_fragment_int[7 + 8 * i]),
                "r"(b_fragment_int[2 + 4 * j]), "r"(b_fragment_int[3 + 4 * j])
            );
        }
    }// end mma compute
    __syncthreads();
    float2 * output_ = reinterpret_cast<float2 *>(Output_Value + (bx * Block_M + warp_x * 64 + (int)(lane_id / 4)) * N + by * Block_N + warp_y * 64 + (int)((lane_id % 4) << 1));
    float2 * output_fragment_ = reinterpret_cast<float2 *>(output_fragment);// 128 reg --> 64 float2
    #pragma unroll
    for(int i = 0; i < 4; i++){
        #pragma unroll
        for(int j = 0; j < 8; j++){
            *(output_ + (i*8  ) * N + j * 4) = *(output_fragment_ + 2 * (i*8+j)    );
            *(output_ + (i*8+4) * N + j * 4) = *(output_fragment_ + 2 * (i*8+j) + 1);
        }
    }
}

hipError_t GEMMex(
    int M, int K, int N,
    const e4m3 * __restrict__ A_Value,
    const e5m2 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    const int Block_M = 128, Block_N = 128, Block_K = 64;
    dim3 block_dim(128,1,1);
	dim3 grid_dim(ceil(static_cast<float>(M) / Block_M), ceil(static_cast<float>(N) / Block_N), 1);

    GEMM_e4m3_e5m2_o32_stage2_row_col<<<grid_dim, block_dim>>>(
        M, K, N, A_Value, B_Value, Output_Value);
	return hipGetLastError();
}
hipError_t GEMM(
    int M, int K, int N,
    const e4m3 * __restrict__ A_Value,
    const e5m2 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
	return GEMMex(M, K, N, A_Value, B_Value, Output_Value);
}
//kernel7////////////////////e5m2 * e4m3 = f32//////////////////
__global__ void GEMM_e5m2_e4m3_o32_stage2_row_col(
    int M, int K, int N,
    const e5m2 * __restrict__ A_Value,
    const e4m3 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    // thread num = 128
    constexpr int Block_M = 128;
    constexpr int Block_K = 64;
    constexpr int Block_N = 128;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int lane_id = threadIdx.x % 32;
    int wid = tid >> 5;// 128 / 32 = 4

    if(bx >= M / Block_M || by >= N / Block_N){
        return;
    }
    constexpr int APAD = 0;
    constexpr int BPAD = 0;

    // extern __shared__ float4 smem[];
    constexpr int smem_a_offset = Block_M * (Block_K + APAD);
    constexpr int smem_b_offset = Block_N * (Block_K + BPAD);
    const int smem_size_float4 = 2 * (smem_a_offset + smem_b_offset) / 16;
    __shared__ float4 smem[smem_size_float4];
    e5m2 * smem_a = reinterpret_cast<e5m2 *>(smem);
    e4m3 * smem_b = reinterpret_cast<e4m3 *>(smem) + 2 * smem_a_offset;

    // __align__(32) float4 matrix_a_fragment[8];// 8 float4 = 32 reg
    // __align__(32) float4 matrix_b_fragment[8];// 8 float4 = 32 reg
    // __align__(32) float output_fragment[128];// 4(m) * 8(n) * 4(one tile) = 128 float
    float4 matrix_a_fragment[8];// 8 float4 = 32 reg
    float4 matrix_b_fragment[8];// 8 float4 = 32 reg
    float output_fragment[128];// 4(m) * 8(n) * 4(one tile) = 128 float

    int smem_a_base_addr = __cvta_generic_to_shared(smem_a);
    int smem_b_base_addr = __cvta_generic_to_shared(smem_b);
    int smem_a_m = (tid / 4) << 2;
    int smem_a_k = (tid &  3) << 4;
    int smem_b_k = (tid &  3) << 4;
    int smem_b_n = (tid / 4) << 2;
    // block_size = 128 * 64 
    // thred num = 128 
    // each thread load = 128 * 64 / 128 = 64 fp8 = 4 float4
    int smem_a_addr0 = smem_a_base_addr + (smem_a_m * (Block_K + APAD) + smem_a_k) * sizeof(char);
    int smem_a_addr1 = smem_a_addr0 + 1 * (Block_K + APAD) * sizeof(char);
    int smem_a_addr2 = smem_a_addr0 + 2 * (Block_K + APAD) * sizeof(char);
    int smem_a_addr3 = smem_a_addr0 + 3 * (Block_K + APAD) * sizeof(char);
    
    int smem_b_addr0 = smem_b_base_addr + (smem_b_n * (Block_K + BPAD) + smem_b_k) * sizeof(char);
    int smem_b_addr1 = smem_b_addr0 + 1 * (Block_K + BPAD) * sizeof(char);
    int smem_b_addr2 = smem_b_addr0 + 2 * (Block_K + BPAD) * sizeof(char);
    int smem_b_addr3 = smem_b_addr0 + 3 * (Block_K + BPAD) * sizeof(char);

    int gmem_a_m = bx * Block_M + smem_a_m;
    int gmem_a_k = smem_a_k;
    int gmem_b_k = smem_b_k;
    int gmem_b_n = by * Block_N + smem_b_n;

    int gmem_a_addr = gmem_a_m * K + gmem_a_k;
    int gmem_b_addr = gmem_b_n * K + gmem_b_k;

    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr0), "l"(&A_Value[gmem_a_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr1), "l"(&A_Value[gmem_a_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr2), "l"(&A_Value[gmem_a_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_a_addr3), "l"(&A_Value[gmem_a_addr + 3 * K]));

    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr0), "l"(&B_Value[gmem_b_addr        ]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr1), "l"(&B_Value[gmem_b_addr + 1 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr2), "l"(&B_Value[gmem_b_addr + 2 * K]));
    asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
        : "r"(smem_b_addr3), "l"(&B_Value[gmem_b_addr + 3 * K]));

    asm ("cp.async.commit_group;\n" ::);
    asm ("cp.async.wait_group 0;\n" ::);
    __syncthreads();

    int warp_x = wid &  1;//0 1 0 1
    int warp_y = wid >> 1;//0 0 1 1
    #pragma unroll 16
    for(int bk = 1; bk < K/Block_K; bk++){
        int sel = (bk & 1) ^ 1; // 0 1 0 1
        int sel_mem = ((bk - 1) & 1) ^ 1; // 1 0 1 0
        gmem_a_addr += Block_K;
        gmem_b_addr += Block_K;
        
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr0 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr1 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 1 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr2 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 2 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_a_addr3 + sel_mem * smem_a_offset * (int)sizeof(char)), "l"(&A_Value[gmem_a_addr + 3 * K]));

        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr0 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr1 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 1 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr2 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 2 * K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(smem_b_addr3 + sel_mem * smem_b_offset * (int)sizeof(char)), "l"(&B_Value[gmem_b_addr + 3 * K]));
        
        float4 * smem_a_sel = reinterpret_cast<float4 *>(smem_a + sel * smem_a_offset + warp_x * (smem_a_offset >> 1));
        float4 * smem_b_sel = reinterpret_cast<float4 *>(smem_b + sel * smem_b_offset + warp_y * (smem_b_offset >> 1));
        matrix_a_fragment[0] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
        matrix_a_fragment[1] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
        matrix_a_fragment[2] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
        matrix_a_fragment[3] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
        matrix_a_fragment[4] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
        matrix_a_fragment[5] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
        matrix_a_fragment[6] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
        matrix_a_fragment[7] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);
        matrix_b_fragment[0] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
        matrix_b_fragment[1] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
        matrix_b_fragment[2] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
        matrix_b_fragment[3] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
        matrix_b_fragment[4] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
        matrix_b_fragment[5] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
        matrix_b_fragment[6] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
        matrix_b_fragment[7] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);

        int * a_fragment_int = reinterpret_cast<int *>(matrix_a_fragment);// 8 float4 --> 32 reg
        int * b_fragment_int = reinterpret_cast<int *>(matrix_b_fragment);// 8 float4 --> 32 reg

        #pragma unroll
        for(int i = 0; i < 4; i++){
            #pragma unroll
            for(int j = 0; j < 8; j++){
                asm ("mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e4m3.f32 \t"
                    "{%0, %1, %2, %3}, \t"
                    "{%4, %5, %6, %7}, \t"
                    "{%8, %9}, \t"
                    "{%0, %1, %2, %3}; ":
                    "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                    "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                    "r"(a_fragment_int[0 + 8 * i]), "r"(a_fragment_int[1 + 8 * i]),
                    "r"(a_fragment_int[4 + 8 * i]), "r"(a_fragment_int[5 + 8 * i]),
                    "r"(b_fragment_int[0 + 4 * j]), "r"(b_fragment_int[1 + 4 * j])
                );
                asm ("mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e4m3.f32 \t"
                    "{%0, %1, %2, %3}, \t"
                    "{%4, %5, %6, %7}, \t"
                    "{%8, %9}, \t"
                    "{%0, %1, %2, %3}; ":
                    "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                    "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                    "r"(a_fragment_int[2 + 8 * i]), "r"(a_fragment_int[3 + 8 * i]),
                    "r"(a_fragment_int[6 + 8 * i]), "r"(a_fragment_int[7 + 8 * i]),
                    "r"(b_fragment_int[2 + 4 * j]), "r"(b_fragment_int[3 + 4 * j])
                );
            }
        }// end mma compute
        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);
        __syncthreads();

    }// end main loop
    int sel = ((K / Block_K) & 1) ^ 1;
    float4 * smem_a_sel = reinterpret_cast<float4 *>(smem_a + sel * smem_a_offset + warp_x * (smem_a_offset >> 1));
    float4 * smem_b_sel = reinterpret_cast<float4 *>(smem_b + sel * smem_b_offset + warp_y * (smem_b_offset >> 1));
    matrix_a_fragment[0] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
    matrix_a_fragment[1] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
    matrix_a_fragment[2] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
    matrix_a_fragment[3] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
    matrix_a_fragment[4] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
    matrix_a_fragment[5] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
    matrix_a_fragment[6] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
    matrix_a_fragment[7] = *(smem_a_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);
    matrix_b_fragment[0] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 0 ) * 4);
    matrix_b_fragment[1] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 8 ) * 4);
    matrix_b_fragment[2] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 16) * 4);
    matrix_b_fragment[3] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 24) * 4);
    matrix_b_fragment[4] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 32) * 4);
    matrix_b_fragment[5] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 40) * 4);
    matrix_b_fragment[6] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 48) * 4);
    matrix_b_fragment[7] = *(smem_b_sel + (lane_id % 4) + (lane_id / 4 + 56) * 4);

    int * a_fragment_int = reinterpret_cast<int *>(matrix_a_fragment);// 8 float4 --> 32 reg
    int * b_fragment_int = reinterpret_cast<int *>(matrix_b_fragment);// 8 float4 --> 32 reg

    #pragma unroll
    for(int i = 0; i < 4; i++){
        #pragma unroll
        for(int j = 0; j < 8; j++){
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e4m3.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[0 + 8 * i]), "r"(a_fragment_int[1 + 8 * i]),
                "r"(a_fragment_int[4 + 8 * i]), "r"(a_fragment_int[5 + 8 * i]),
                "r"(b_fragment_int[0 + 4 * j]), "r"(b_fragment_int[1 + 4 * j])
            );
            asm ("mma.sync.aligned.m16n8k32.row.col.f32.e5m2.e4m3.f32 \t"
                "{%0, %1, %2, %3}, \t"
                "{%4, %5, %6, %7}, \t"
                "{%8, %9}, \t"
                "{%0, %1, %2, %3}; ":
                "+f"(output_fragment[0 + 4 * (8*i+j)]), "+f"(output_fragment[1 + 4 * (8*i+j)]),
                "+f"(output_fragment[2 + 4 * (8*i+j)]), "+f"(output_fragment[3 + 4 * (8*i+j)]):
                "r"(a_fragment_int[2 + 8 * i]), "r"(a_fragment_int[3 + 8 * i]),
                "r"(a_fragment_int[6 + 8 * i]), "r"(a_fragment_int[7 + 8 * i]),
                "r"(b_fragment_int[2 + 4 * j]), "r"(b_fragment_int[3 + 4 * j])
            );
        }
    }// end mma compute
    __syncthreads();
    float2 * output_ = reinterpret_cast<float2 *>(Output_Value + (bx * Block_M + warp_x * 64 + (int)(lane_id / 4)) * N + by * Block_N + warp_y * 64 + (int)((lane_id % 4) << 1));
    float2 * output_fragment_ = reinterpret_cast<float2 *>(output_fragment);// 128 reg --> 64 float2
    #pragma unroll
    for(int i = 0; i < 4; i++){
        #pragma unroll
        for(int j = 0; j < 8; j++){
            *(output_ + (i*8  ) * N + j * 4) = *(output_fragment_ + 2 * (i*8+j)    );
            *(output_ + (i*8+4) * N + j * 4) = *(output_fragment_ + 2 * (i*8+j) + 1);
        }
    }
}

hipError_t GEMMex(
    int M, int K, int N,
    const e5m2 * __restrict__ A_Value,
    const e4m3 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    const int Block_M = 128, Block_N = 128, Block_K = 64;
    dim3 block_dim(128,1,1);
	dim3 grid_dim(ceil(static_cast<float>(M) / Block_M), ceil(static_cast<float>(N) / Block_N), 1);

    GEMM_e5m2_e4m3_o32_stage2_row_col<<<grid_dim, block_dim>>>(
        M, K, N, A_Value, B_Value, Output_Value);
	return hipGetLastError();
}
hipError_t GEMM(
    int M, int K, int N,
    const e5m2 * __restrict__ A_Value,
    const e4m3 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
	return GEMMex(M, K, N, A_Value, B_Value, Output_Value);
}

//kernel4////////////////////e5m2*e5m2 = f32//////////////////
__global__ void GEMM_e5m2_e5m2_o32_stage4_row_col(
    int M, int K, int N,
    const e5m2 * __restrict__ A_Value,
    const e5m2 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
}
hipError_t GEMMex4(
    int M, int K, int N,
    const e5m2 * __restrict__ A_Value,
    const e5m2 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    const int Block_M = 128, Block_N = 128, Block_K = 64;
    dim3 block_dim(128,1,1);
	dim3 grid_dim(ceil(static_cast<float>(M) / Block_M), ceil(static_cast<float>(N) / Block_N), 1);
    // unsigned int dsmem = 4 * (Block_M * (Block_K + 0) + Block_N * (Block_K + 0)) * sizeof(char);

    GEMM_e5m2_e5m2_o32_stage4_row_col<<<grid_dim, block_dim>>>(
        M, K, N, A_Value, B_Value, Output_Value);
	return hipGetLastError();
}
hipError_t GEMM4(
    int M, int K, int N,
    const e5m2 * __restrict__ A_Value,
    const e5m2 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
	return GEMMex(M, K, N, A_Value, B_Value, Output_Value);
}
//kernel6////////////////////e4m3*e5m2 = f32//////////////////
__global__ void GEMM_e4m3_e5m2_o32_stage4_row_col(
    int M, int K, int N,
    const e4m3 * __restrict__ A_Value,
    const e5m2 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
}
hipError_t GEMMex4(
    int M, int K, int N,
    const e4m3 * __restrict__ A_Value,
    const e5m2 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    const int Block_M = 128, Block_N = 128, Block_K = 64;
    dim3 block_dim(128,1,1);
	dim3 grid_dim(ceil(static_cast<float>(M) / Block_M), ceil(static_cast<float>(N) / Block_N), 1);
    // unsigned int dsmem = 4 * (Block_M * (Block_K + 0) + Block_N * (Block_K + 0)) * sizeof(char);

    GEMM_e4m3_e5m2_o32_stage4_row_col<<<grid_dim, block_dim>>>(
        M, K, N, A_Value, B_Value, Output_Value);
	return hipGetLastError();
}
hipError_t GEMM4(
    int M, int K, int N,
    const e4m3 * __restrict__ A_Value,
    const e5m2 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
	return GEMMex(M, K, N, A_Value, B_Value, Output_Value);
}
//kernel8////////////////////e5m2*e4m3 = f32//////////////////
__global__ void GEMM_e5m2_e4m3_o32_stage4_row_col(
    int M, int K, int N,
    const e5m2 * __restrict__ A_Value,
    const e4m3 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
}
hipError_t GEMMex4(
    int M, int K, int N,
    const e5m2 * __restrict__ A_Value,
    const e4m3 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
    const int Block_M = 128, Block_N = 128, Block_K = 64;
    dim3 block_dim(128,1,1);
	dim3 grid_dim(ceil(static_cast<float>(M) / Block_M), ceil(static_cast<float>(N) / Block_N), 1);
    // unsigned int dsmem = 4 * (Block_M * (Block_K + 0) + Block_N * (Block_K + 0)) * sizeof(char);

    GEMM_e5m2_e4m3_o32_stage4_row_col<<<grid_dim, block_dim>>>(
        M, K, N, A_Value, B_Value, Output_Value);
	return hipGetLastError();
}
hipError_t GEMM4(
    int M, int K, int N,
    const e5m2 * __restrict__ A_Value,
    const e4m3 * __restrict__ B_Value,
    float * __restrict__ Output_Value)
{
	return GEMMex(M, K, N, A_Value, B_Value, Output_Value);
}
}